#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

int *a, *b;  // host data
int *c;  // result

//Cuda error checking - non mandatory
void cudaCheckError() {
 hipError_t e=hipGetLastError();
 if(e!=hipSuccess) {
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
   exit(0); 
 }
}

//GPU kernel 
__global__
void multiplicationCPU(int *A,int *B,int *C,int N){
    //Get current column and row
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int element = 0;
    //Verify if it's in the bounds
    if (row < N && col < N){
        //Dot product to compute and the element C[col, row]
        for (int i = 0; i < N; ++i)
        {
            element += A[row*N + i] * B[i*N + col];

        }
    }
    //Store result of the element
    C[row*N + col] = element;
}



int main(int argc,char **argv)
{
    printf("Begin \n");
    //Iterations
    int n=4;
    //Number of blocks (4x4)
    int nBytes = n*n*sizeof(int);
    //Block size and number
    int block_size, block_no;

    //memory allocation 
    a = (int *) malloc(nBytes);
    b = (int *) malloc(nBytes);
    c = (int *) malloc(nBytes);

    int *a_d,*b_d,*c_d;
    block_size = n; //threads per block
    block_no = n/block_size;
    
    //Work definition
    dim3 dimBlock(block_size, block_size);
    dim3 dimGrid(block_no, block_no);

    // Data filling
    for(int i=0;i<n*n;i++)
    {
        a[i]=i;
        b[i]=i;
    }
        

    //Showing data
    printf("Matrix A:\n");
    for (int i = 0; i < 4; ++i)
    {
        for (int j = 0; j < 4; ++j)
        {
            printf("%d\t", a[i*4+j]);
        }
        printf("\n");
    }
    printf("\nMatrix B:\n");
    for (int i = 0; i < 4; ++i)
    {
        for (int j = 0; j < 4; ++j)
        {
            printf("%d\t", b[i*4+j]);
        }
        printf("\n");
    }

    printf("\n\nAllocating device memory on host..\n");
   //GPU memory allocation
    hipMalloc((void **) &a_d, n*n*sizeof(int));
    hipMalloc((void **) &b_d, n*n*sizeof(int));
    hipMalloc((void **) &c_d, n*n*sizeof(int));

    printf("Copying to device..\n");
    hipMemcpy(a_d, a, n*n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, n*n*sizeof(int), hipMemcpyHostToDevice);

    //Starting clock
    clock_t start_d=clock();
    printf("Doing GPU matrix multiplicationCPU\n\n");
    multiplicationCPU<<<dimGrid,dimBlock>>>(a_d, b_d, c_d, n);
    cudaCheckError();

    //Wait for kernel call to finish
    hipDeviceSynchronize();

    clock_t end_d = clock();
    
    //Time computing
    double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;

    //Copying data back to host, this is a blocking call and will not start until all kernels are finished
    hipMemcpy(c, c_d, n*n*sizeof(int), hipMemcpyDeviceToHost);
    printf("n = %dx%d \t GPU time = %fs \n\n", n, n, time_d);

    //Showing result
    printf("Matrix C:\n");
    for (int i = 0; i < 4; ++i)
    {
        for (int j = 0; j < 4; ++j)
        {
            printf("%d\t", c[i*4+j]);
        }
        printf("\n");
    }

    //Free GPU memory
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    return 0;
}